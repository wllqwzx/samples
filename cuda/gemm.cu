#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void gemm(const float *a, const float *b, float *c, const size_t m,
                     const size_t n, const size_t k) {
  int i = blockIdx.x;
  int j = threadIdx.x;
  float acc = 0;
  for (size_t t = 0; t < k; ++t) {
    acc += a[i * 1024 + t] * b[j * 1024 + t];
  }
  c[i * 1024 + j] = acc;
}

int main() {
  const size_t m = 1024;
  const size_t n = 1024;
  const size_t k = 1024;
  const size_t n_iter = 100;

  float *host_a, *host_b, *host_c;
  hipHostMalloc(&host_a, m * k * sizeof(float));
  hipHostMalloc(&host_b, n * k * sizeof(float));
  hipHostMalloc(&host_c, m * n * sizeof(float));
  random_init<float>(host_a, m * k);
  random_init<float>(host_b, n * k);

  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, m * k * sizeof(float));
  hipMalloc(&d_b, n * k * sizeof(float));
  hipMalloc(&d_c, m * n * sizeof(float));

  hipMemcpy(d_a, host_a, m * k * sizeof(float), hipMemcpyDefault);
  hipMemcpy(d_b, host_b, n * k * sizeof(float), hipMemcpyDefault);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  // warmup
  for (size_t i = 0; i < 10; ++i) {
    gemm<<<1024, 1024>>>(d_a, d_b, d_c, m, n, k);
  }

  hipEventRecord(start);
  for (size_t i = 0; i < n_iter; ++i) {
    gemm<<<1024, 1024>>>(d_a, d_b, d_c, m, n, k);
  }
  hipEventRecord(end);
  hipEventSynchronize(end);

  float ms;
  hipEventElapsedTime(&ms, start, end);

  hipEventDestroy(start);
  hipEventDestroy(end);

  size_t flop = m * n * k * 2;
  double gflops = (double(flop) / 1e9) / ((double(ms) / n_iter) / 1e3);
  printf("GFlops: %f GFlops\n", gflops);

  hipMemcpy(host_c, d_c, m * n * sizeof(float), hipMemcpyDefault);

  // TODO: check correctness of host_c

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipHostFree(host_a);
  hipHostFree(host_b);
  hipHostFree(host_c);
  return 0;
}
